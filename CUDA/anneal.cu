#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <iostream>
#include <hip/hip_runtime.h>

#define MAX_TRIES 10
#define N_LIMIT 20
#define MAX_TEMP_STEPS 500
#define TEMP_START 0.5
#define COOLING 0.99
#define THREADS 1024
#define MAX_CITY 1024
#define BOLTZMANN_COEFF 0.01

#define CUDA_CALL(x) do {if((x) != hipSuccess) {\
	printf("Error at %s:%d\n",__FILE__,__LINE__); \
	return EXIT_FAILURE;}} while(0)

using namespace std;

struct city {
	double x;
	double y;
};

struct permutation {
	int cost;
	int order[MAX_CITY];
	int nSucc;
};

struct GlobalConstants {
	int CITY_N;
	city* cities;
	unsigned int* randSeeds;
};

//global variables
struct city *cities;
int CITY_N;

//global variables on GPU
__constant__ GlobalConstants cuTspParam;


/* rounding function, but at .5 rounds to the lower int. Due to the TSPLIB
 * standard library.
 */
__device__ __host__ int nint(float x)
{
	return (int) (x + 0.5);
}
/* Randomisation is done by a simple linear congruential generator.
 * We use A and C values as done by glibc.
 */

__device__ unsigned int randomInt(hiprandState *state, unsigned int max) {
	return hiprand(state) % max;
}

__device__ double randomDouble(hiprandState *state)
{
	return (double) hiprand_uniform(state);
}

__device__ bool randomBool(hiprandState *state)
{
	if ((randomInt(state, 256) >> 7) & 0x00000001)
		return true;
	else
		return false;
}

__global__ void initCurand(hiprandState *state, unsigned long seed) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(seed, idx, 0, &state[idx]);
}

__device__ __host__ int euclideanDistance(struct city *a, struct city *b)
{
	float dx = b->x - a->x;
	float dy = b->y - a->y;
	return nint((sqrt(dx * dx + dy * dy)));
}

/* Calcuates the delta of the costs given by a new order using reverse
 */
__device__ int reverseCost(struct city *cities, int *order, int *n)
{
	int cost;

	cost = -euclideanDistance(&cities[order[n[0]]], &cities[order[n[2]]]);
	cost -= euclideanDistance(&cities[order[n[1]]], &cities[order[n[3]]]);
	cost += euclideanDistance(&cities[order[n[0]]], &cities[order[n[3]]]);
	cost += euclideanDistance(&cities[order[n[1]]], &cities[order[n[2]]]);

	return cost;
}

/* The order of the city is changed by swapping the 
 * order between n[0] and n[1]. 
 * The swapping is done beginning from the outer end
 * going into the middle
 */
__device__ void reverse(int *order, int *n)
{
	int swaps = (1 + ((n[1] - n[0] + CITY_N) % CITY_N)) / 2;	// this many elements have to be swapped to have a complete reversal
	for (int j = 0; j < swaps; ++j) {
		int k = (n[0] + j) % CITY_N;
		int l = (n[1] - j + CITY_N) % CITY_N;
		int tmp = order[k];
		order[k] = order[l];
		order[l] = tmp;
	}
}

/* Calculates the delta of the costs of the city order if
 * the transportation of this segments (given by n) are actually
 * done.
 */
__device__ int transportCost(struct city *cities, int *order, int *n)
{
	int cost;

	cost = -euclideanDistance(&cities[order[n[1]]], &cities[order[n[5]]]);
	cost -= euclideanDistance(&cities[order[n[0]]], &cities[order[n[4]]]);
	cost -= euclideanDistance(&cities[order[n[2]]], &cities[order[n[3]]]);
	cost += euclideanDistance(&cities[order[n[0]]], &cities[order[n[2]]]);
	cost += euclideanDistance(&cities[order[n[1]]], &cities[order[n[3]]]);
	cost += euclideanDistance(&cities[order[n[4]]], &cities[order[n[5]]]);

	return cost;
}

/* Transport the path segment (consisting of the start n[0] and end at n[1]
 * to the path given by n[2] and n[3], which are adjacent and the segment is
 * to be placed in between. n[4] is the city preceding n[0] and n[5] succeeds
 * n[1].
 * Transportation should only be done if the metroplis algorithm agrees.
 *
 */
__device__ void transport(int *order, int *n)
{
	int newOrder[MAX_CITY];
	int m1 = (n[1] - n[0] + CITY_N) % CITY_N;
	int m2 = (n[4] - n[3] + CITY_N) % CITY_N;
	int m3 = (n[2] - n[5] + CITY_N) % CITY_N;
	int i = 0;
	for (int j = 0; j <= m1; ++j) {
		newOrder[i++] = order[(j + n[0]) % CITY_N];
	}
	for (int j = 0; j <= m2; ++j) {
		newOrder[i++] = order[(j + n[3]) % CITY_N];
	}
	for (int j = 0; j <= m3; ++j) {
		newOrder[i++] = order[(j + n[5]) % CITY_N];
	}
	for (int j = 0; j < CITY_N; ++j) {
		order[j] = newOrder[j];
	}
}

/* Metroplis algorithm: Always take the downhill path and
 * sometime take the uphill path to avoid local minima
 */
__device__ bool metropolis(const int cost, const double t, unsigned int *x)
{
	return cost < 0 || randomDouble(x) < exp((double) (BOLTZMANN_COEFF * -cost / t));
}

/* Main kernel function */
__global__ void solve(struct permutation *permutations, const float t)
{
	struct city* cities = cuTspParam.cities;
	int CITY_N = cuTspParam.CITY_N;
	int notSeg;						// number of cities not on the segment
	int maxChangeTries = MAX_TRIES * CITY_N;
	int succLimit = N_LIMIT * CITY_N;
	int dCost;
	bool ans;
	int n[6];
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	struct permutation *perm = &(permutations[id]);
	unsigned int *x = &(lcg_x[id]);

	perm->nSucc = 0;
	for (int j = 0; j < maxChangeTries; ++j) {
		do {
			n[0] = randomInt(x, CITY_N);
			n[1] = randomInt(x, CITY_N - 1);
			if (n[1] >= n[0]) 
				++n[1];
			notSeg = (n[0] - n[1] + CITY_N - 1) % CITY_N;
		} while (notSeg < 2);

		/* It is randomly choosen whether a transportation or a reversion is done */
		if (randomBool(x)) {
			n[2] = (n[1] + randomInt(x, abs(notSeg - 1)) + 1) % CITY_N;
			n[3] = (n[2] + 1) % CITY_N;
			n[4] = (n[0] + CITY_N- 1) % CITY_N;
			n[5] = (n[1] + 1) % CITY_N;

			dCost = transportCost(cities, perm->order, n);
			ans = metropolis(dCost, t, x);
			if (ans) {
				++perm->nSucc;
				perm->cost += dCost;
				transport(perm->order, n);
			}
		} else {
			n[2] = (n[0] + CITY_N - 1) % CITY_N;
			n[3] = (n[1] + 1) % CITY_N;

			dCost = reverseCost(cities, perm->order, n);
			ans = metropolis(dCost, t, x);
			if (ans) {
				++perm->nSucc;
				perm->cost += dCost;
				reverse(perm->order, n);
			}
		}

		/* Finish early if there are enough successful changes */
		if (perm->nSucc > succLimit)
			break;
	}
}

class Anneal {
private:
	/* Calculates the length of the initial path, which is already given.
	 * This is in O(n)
	 */
	void initialPath(struct permutation *perm, struct city *cities)
	{
		int i, i1, i2;

		perm->cost= 0;
		for (i = 0; i < CITY_N - 1; i++) {
			i1 = perm->order[i];
			i2 = perm->order[i+1];
			perm->cost += euclideanDistance(&cities[i1], &cities[i2]);
		}
		i1 = perm->order[CITY_N - 1];
		i2 = perm->order[0];
		perm->cost += euclideanDistance(&cities[i1], &cities[i2]);
		cout << "Initial path length: " << perm->cost << endl;
	}

	void printInformation(struct permutation *currPerm, bool showOrder = true)
	{
		cout << "Path Length = " << currPerm->cost << endl;
		cout << "Successful Moves: " << currPerm->nSucc << endl;
		if (showOrder) {
			cout << "Order: ";
			for (int j = 0; j < CITY_N; j++) {
				cout << currPerm->order[j] << " ";
			}
		}
		cout << endl;
	}

public:
	double runtime;
	int resultCost;

	Anneal() {}

	void order(struct city *cities, int *order)
	{
		double t = TEMP_START;
		hipError_t cudaStat;
		struct permutation *dPermutation;
		struct permutation *hPermutation = (struct permutation *) malloc(THREADS * sizeof(struct permutation));
		struct city *dCities;
		struct permutation *currPerm = (struct permutation *) malloc(sizeof(struct permutation));
		struct permutation *allMinPerm= (struct permutation *) malloc(sizeof(struct permutation));
		int oldCost = 2147483647;
		int repeatCost = 0;
		clock_t startAll, endAll;			// timer to measure the overall run time
		double runtimeAll;
		clock_t startCuda, endCuda;			//timer to measure the run time of cuda
		double cudaRuntime = 0.0f;
		hiprandState *devStates;

		startAll = clock();

		// Kernel invocation
		int threadsPerBlock = 256;
		int blocksPerGrid = (THREADS + threadsPerBlock - 1) / threadsPerBlock;

		cout << "Threads: " << THREADS << ", Blocks: " << blocksPerGrid << endl;

		memcpy(currPerm->order, order, CITY_N * sizeof(int));
		initialPath(currPerm, cities);
		memcpy(allMinPerm, currPerm, sizeof(struct permutation));

		CUDA_CALL(hipMalloc(&dPermutation, THREADS * sizeof(struct permutation)));
		CUDA_CALL(hipMalloc(&dCities, CITY_N * sizeof(struct city)));
		CUDA_CALL(hipMemcpy(dCities, cities, CITY_N * sizeof(struct city), hipMemcpyHostToDevice));

		// for generate random numbers directly on the device
		CUDA_CALL(hipMalloc(void **)&devStates, THREADS * sizeof(hiprandState));
		initCurand<<<blocksPerGrid, threadsPerBlock>>>(devStates, 1234);
		
		/* Try up to MAX_TEMP_STEPS temperature steps. It could stop before if no kernel
		 * showed any succesful change or if the solution did not change 5 times
		 */
		for (int i = 0; i < MAX_TEMP_STEPS; ++i) {
			hipDeviceSynchronize();
			startCuda = clock();

			//Copies the initial permutation to each result permutation
			for (int i = 0; i < THREADS; ++i) {
				memcpy(hPermutation[i].order, currPerm->order, CITY_N * sizeof(int));
				hPermutation[i].cost = currPerm->cost;
			}
			cudaStat = hipMemcpy(dPermutation, hPermutation, THREADS * sizeof(struct permutation), hipMemcpyHostToDevice);
			if (cudaStat != hipSuccess) {
				cout << "couldn't copy memory to global memory. Exit." << endl;
				return;
			}

			//invoke cuda
			solve<<<blocksPerGrid, threadsPerBlock>>>(dPermutation, t);

			cudaStat = hipDeviceSynchronize();
			if (cudaStat != hipSuccess) {
				cout << "something went wrong during device execution. Exit." << endl;
				return;
			}

			endCuda = clock();
			cudaRuntime += (endCuda - startCuda) * 1000 / CLOCKS_PER_SEC;

			cudaStat = hipMemcpy(hPermutation, dPermutation, THREADS * sizeof(struct permutation), hipMemcpyDeviceToHost);
			if (cudaStat != hipSuccess) {
				cout << "couldn't copy memory from global memory. Exit." << endl;
				return;
			}

			/* Loops through all resulting permutations and store the one with minimal length but
			 * at least one swap.
			 * If all threads didn't swap, exit the program.
			 * Takes O(n) time. 
			 */
			int minCost = 2147483647;
			bool swap = false;
			for (int j = 0; j < THREADS; ++j) {
				if (minCost >= hPermutation[j].cost && hPermutation[j].nSucc != 0) {
					currPerm = &(hPermutation[j]);
					minCost = currPerm->cost;
					swap = true;
					if (minCost < allMinPerm->cost)
						memcpy(allMinPerm, currPerm, sizeof(struct permutation));
				}
			}

			if (!swap) {
				cout << "No swaps occured. Exit" << endl;
				break;
			}

			if (oldCost == minCost) {
				if (++repeatCost == 5) {
					cout << "Cost did not change 5 times in a row. Exit" << endl;
					break;
				}
			} else
				repeatCost = 0;

			cout << endl << "T = " <<  t << endl;
			//cout << "repeat: " << repeatCost << ", old: " << oldCost << ", new: " << minCost << endl;
			printInformation(currPerm, false);
			//for (int j = 0; j < THREADS; ++j)
			//	printInformation(&(hPermutation[j]), false);

			oldCost = minCost;
			t *= COOLING;
		}

		endAll = clock();
		runtimeAll = (endAll - startAll) / (1.0f * CLOCKS_PER_SEC) * 1000;

		cout << endl << "Final Result:" << endl;
		cout << "=============" << endl;
		printInformation(allMinPerm);

		runtime = runtimeAll;
		resultCost = allMinPerm->cost;

		printf("\nThe program needed an overall time of %.2lf ms.\n", runtimeAll);
		printf("%.2lf ms were spent at the CUDA part.\n", cudaRuntime);
		printf("So %.2lf ms were spent at the host.", runtimeAll - cudaRuntime);

		hipFree(dPermutation);
		hipFree(dCities);

		free(allMinPerm);
		free(hPermutation);
	}
};
